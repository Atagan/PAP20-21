#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void sumarMatriz(int *a, int *b, int N) { //a es una matriz, b la otra, N es el numero de elementos total del array que representa la matriz.
	int idThread = blockIdx.x * blockDim.x + threadIdx.x; //comprobamos que no haya ningun problema

	if (idThread < N) { //Esta comprobacion existe por si acaso, idx nunca deberia superar a N
		a[idThread] = a[idThread] + b[idTread];
	}
}


int main() {
	hipError_t error = hipSuccess;

	//reservamos memoria en el host para el vector
	size_t size = 4 * 4 sizeof(int);
	int* host_matrixA = (int*)malloc(size);
	int* host_matrixB = (int*)malloc(size);


	for (int i = 0; i < 4; i++) {
		for(int j = 0; i < 4; j++)
			host_matrixA[j + (i - 1) * 4] = i + j;
			host_matrixB[j + (i - 1) * 4] = i;
		}
	}

	//reservamos la memoria para el dispositivo AKA: GPU
	int* device_matrixA = nullptr;
	int* device_matrixB = nullptr;

	error = hipMalloc((void**)&device_matrixA, size);

	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}

	error = hipMalloc((void**)&device_matrixB, size);

	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}

	//Copiar memoria de host a device
	error = hipMemcpy(device_matrixA, host_matrixA, size, hipMemcpyHostToDevice);

	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}
	
	error = hipMemcpy(device_matrixB, host_matrixB, size, hipMemcpyHostToDevice);

	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}

	//Lanzar el kernel que haga la operacion

	int threadsPerBlock = 256;
	int blocksPerGrid = (16 + threadsPerBlock - 1) / threadsPerBlock;
	sumarMatriz <<<blocksPerGrid, threadsPerBlock >>> (device_matrixA, device_matrixB, 16);


	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}

	//Copiamos la memoria del device al host
	printf("El vector original: [%d, %d, %d, %d, %d, %d, %d, %d]\n", host_vector[0], host_vector[1], host_vector[2], host_vector[3], host_vector[4], host_vector[5], host_vector[6], host_vector[7]);

	error = hipMemcpy(host_matrixA, device_matrixA, size, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}

	printf("El vector tras sumarle 10: [%d, %d, %d, %d, %d, %d, %d, %d]\n", host_vector[0], host_vector[1], host_vector[2], host_vector[3], host_vector[4], host_vector[5], host_vector[6], host_vector[7]);

	
	error = hipFree(device_matrixA);
	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}
	error = hipFree(device_matrixB);
	if (error != hipSuccess) {
		printf("addWithCuda failed!");
		return 1;
	}
	free(host_matrixA);
	free(host_matrixB);

	printf("Ejecucion del programa correcta");
	return(0);
}
